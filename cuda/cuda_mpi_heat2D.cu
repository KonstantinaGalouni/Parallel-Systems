#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <lcutil.h>
#include <timestamp.h>

#define NXPROB      3600                 /* x dimension of problem grid */
#define NYPROB      3600  
#define STEPS       1000


void inidat(int nx, int ny, float *u) {
int ix, iy;

for (ix = 0; ix <= nx-1; ix++) 
  for (iy = 0; iy <= ny-1; iy++)
     *(u+ix*ny+iy) = (float)(ix * (nx - ix - 1) * iy * (ny - iy - 1));
}
__global__ void update(int, float*, float*);



extern "C" void updateGPU()
{
  	float *u1,*u2, *host_array;
	int it;
	timestamp t_start;	
	
	/* creating two 1d arrays for cuda */

	CUDA_SAFE_CALL(hipMalloc((void**)&u1,(unsigned long)(NXPROB*NYPROB*sizeof(float))));
	CUDA_SAFE_CALL(hipMalloc((void**)&u2,(unsigned long)(NXPROB*NYPROB*sizeof(float))));

	/* creating host_array to initialiaze */

	host_array=(float*)malloc(NXPROB*NYPROB*sizeof(float));
	memset(host_array,0,NXPROB*NYPROB*sizeof(float));
	
	/* transfering the host_array to device */

	CUDA_SAFE_CALL(hipMemcpy(u2,host_array,NXPROB*NYPROB*sizeof(float),hipMemcpyHostToDevice));

	inidat(NXPROB, NYPROB, host_array);
	
	CUDA_SAFE_CALL(hipMemcpy(u1,host_array,NXPROB*NYPROB*sizeof(float),hipMemcpyHostToDevice));	
	 
	dim3 NumberOfThreads(NXPROB-2);			
	dim3 NumberOfBlocks(NYPROB-2);	
	
	int iz = 0; 
	
	t_start = getTimestamp();
	
	for (it = 1; it <= STEPS; it++)
	{
		/* swapping between the two arrays */
       
		if(iz==0){
			update<<<NumberOfBlocks,NumberOfThreads>>>(NYPROB,u1,u2);
		}else{
			update<<<NumberOfBlocks,NumberOfThreads>>>(NYPROB,u2,u1);
		}		
        iz = 1 - iz;
        		
		CUDA_SAFE_CALL(hipDeviceSynchronize() );		
	}
			
	printf("Time elapsed = %6.4lf in ms\n",getElapsedtime(t_start));
	
	/* Copy results back to host memory */

	CUDA_SAFE_CALL( hipMemcpy(host_array, u2, NXPROB*NYPROB*sizeof(float), hipMemcpyDeviceToHost) );	
	CUDA_SAFE_CALL( hipFree(u1) );	
	CUDA_SAFE_CALL( hipFree(u2) );
	
	
	/*int i,j
	  for(i=0;i<NXPROB;i++){
		for(j=0;j<NYPROB;j++){
			printf("%5.1f ",*(host_array + i*NYPROB + j));
		}
		printf("\n");
	}*/	

    free(host_array);
	
	
}


__global__ void update( int ny, float *u1, float *u2)
{	
	struct Parms { 
		float cx;
		float cy;
	} parms = {0.1, 0.1};
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int ix = idx / (ny-2) + 1;
	int iy = idx % (ny-2) + 1;
 
  	
     *(u2+ix*ny+iy) =  *(u1+ix*ny+iy)  + 
                  	  parms.cx * (*(u1+(ix+1)*ny+iy) +
                      *(u1+(ix-1)*ny+iy) - 
                      2.0 * *(u1+ix*ny+iy)) +
                      parms.cy * (*(u1+ix*ny+iy+1) +
                     *(u1+ix*ny+iy-1) - 
                      2.0 * *(u1+ix*ny+iy));	
	
	__syncthreads();
}

	


int main(int argc,char *argv[])
{
	updateGPU();
}





